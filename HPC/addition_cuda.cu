#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

__global__ void addVect(int *A int *B, int *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(n<100)
    {
        C[i] = A[i]+B[i];
    }
}


int main(){

int n=100;
int *A,*B,*C;
int size = n*sizeof(int);
hipHostMalloc(&A, size);
hipHostMalloc(&B, size);
hipHostMalloc(&C, size);


for(int i=0;i<n;i++)
{
    A[i] = i;
    B[i] = i*2;
}

int *da, *db, *dc;
hipMalloc(&da, size);
hipMalloc(&db, size);
hipMalloc(&dc, size);

hipMemcpy(da, A, size, hipMemcpyHostToDevice);
hipMemcpy(db, B, size, hipMemcpyHostToDevice);

int blocksize = 256;
int numblock = (n+blocksize-1)/blocksize;

addVect<<<numblock, bloacksize>>> (da,db,dc,n);
hipMemcpy(C, dc, size, hipMemcpyDeviceToHost);
hipFree(da);
hipFree(db);
hipFree(dc);
hipHostFree(A);
hipHostFree(B);
hipHostFree(C);
return 0;

}